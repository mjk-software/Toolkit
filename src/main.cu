#include "hip/hip_runtime.h"
/*
* main.cu
* Author: Mateusz Kaleta
*/

#include<iostream>
#include <sys/time.h>

#include "Matrix.h"
#include "Toolkit.h"
#include "CPUToolkit.h"
#include "GPUToolkit.h"

double cpuTime() {
  struct timeval clock;
  gettimeofday(&clock, NULL);
  return ((double)clock.tv_sec + (double)clock.tv_usec *1.e-6);
}

void evaluatePerformance(std::string operation, Toolkit *T, int N,
int matrix_size[], double time[]){
  int range = 10;
  double elapsed_time = 0;
  for(int i=0; i<N; i++){
    if (operation=="add"){
      Matrix A(matrix_size[i], matrix_size[i], range);
      Matrix B(matrix_size[i], matrix_size[i], range);
      elapsed_time = cpuTime();
      Matrix C = T->add(A,B);
      elapsed_time = cpuTime() - elapsed_time;
    }
    if (operation=="subtract"){
      Matrix A(matrix_size[i], matrix_size[i], range);
      Matrix B(matrix_size[i], matrix_size[i], range);
      elapsed_time = cpuTime();
      Matrix C = T->subtract(A,B);
      elapsed_time = cpuTime() - elapsed_time;
    }
    if (operation=="multiply"){
      Matrix A(matrix_size[i], matrix_size[i], range);
      Matrix B(matrix_size[i], matrix_size[i], range);
      elapsed_time = cpuTime();
      Matrix C = T->multiply(A,B);
      elapsed_time = cpuTime() - elapsed_time;
    }
    if (operation=="inverse"){
      Matrix A(matrix_size[i], matrix_size[i], range);
      elapsed_time = cpuTime();
      Matrix C = T->findInverse(A);
      elapsed_time = cpuTime() - elapsed_time;
    }
    time[i] = elapsed_time;
  }
}


int main(int argc, char** argv){
  srand(time(NULL));

  if (argc !=4){
	    std::cerr << "Wrong input parameters. Usage: " << argv[0] << " [gpu|cpu] \
  [add|subtract|multiply|inverse] [normal|performance]" << std::endl;
	    return 1;
  }
  std::string architecture = argv[1];
  std::string operation = argv[2];
  std::string type = argv[3];

  if (  !(architecture == "gpu" || architecture == "cpu")
     || !(operation == "add" || operation == "subtract" || operation == "multiply" || operation == "inverse")
     || !(type == "normal" || type == "performance")
  ){
    std::cerr << "Wrong input parameters. Usage: " << argv[0] << " [gpu|cpu] \
  [add|subtract|multiply|inverse] [normal|performance]" << std::endl;
    return 1;
  }

  Toolkit *T;

  if(architecture == "cpu"){
    T = new CPUToolkit();
    std::cout << "Used architecture: cpu" << std::endl;
  }
  else if (architecture == "gpu"){
    T = new GPUToolkit();
    std::cout << "Used architecture: gpu" << std::endl;
  }
  if (type == "performance"){
    int N = 25;
    int matrix_size[N];
    for (int idx=0; idx<N;idx++) matrix_size[idx] = 4*(idx+1);
    double time[N];
    evaluatePerformance(operation, T, N, matrix_size, time);
    std::cout << "Performed operation: " << operation << std::endl;
    for (int i=0; i<N;i++){
      std::cout << "Matrix size : " << matrix_size[i] << ", " << "time: " << time[i] << " s" << std::endl;
    }
  }
  // two operand operations
  else if (operation == "add" || operation == "subtract" || operation == "multiply"){
    std::string A_path = "./A.txt";
    std::string B_path = "./B.txt";
    // Read matrices from text files
    Matrix A(A_path);
    Matrix B(B_path);
    std::cout << "\nMatrix A: " << A << std::endl;
    std::cout << "\nMatrix B: " << B << std::endl;
    if (operation=="add"){
      Matrix C = T->add(A, B);
      std::cout << "\nMatrix A+B: " << C << std::endl;
    }
    if (operation=="subtract"){
      Matrix C = T->subtract(A, B);
      std::cout << "\nMatrix A-B: " << C << std::endl;
    }
    if (operation=="multiply"){
      Matrix C = T->multiply(A, B);
      std::cout << "\nMatrix A*B: " << C << std::endl;
    }
  }
  // one operand operations
  else if (operation == "inverse"){
    std::string A_path = "./A.txt";
    // Read one matrix from text file
    Matrix A(A_path);
    std::cout << "\nMatrix A: " << A << std::endl;
    Matrix C = T->findInverse(A);
    std::cout << "\nA^(-1): " << C << std::endl;
  }
  delete T;
}
