#include "hip/hip_runtime.h"
/*
* GPUToolkit.h
* Author: Mateusz Kaleta
*/


#include "GPUToolkit.h"
#include "Matrix.h"

#define BLOCK_SIZE 16


__global__ void find_pivots(double *d_a, double *d_id, int n, int row){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  double eps = 1e-6;
  if(col < n){
    if ( abs(d_a[row*n+row]) < eps)
    for(int k=row+1; k<n; k++){
      if ( abs(d_a[k*n+row]) < eps){
        d_a[row*n+col] += d_a[k*n+col];
        d_id[row*n+col] += d_id[k*n+col];
        break;
      }
    }
  }
}

__global__ void normalize_row(double *d_a, double *d_id, int n, int row){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < n){
    if (row !=col){
      d_id[row*n+col] /= d_a[row*n+row];
      d_a[row*n+col] /= d_a[row*n+row];
    }
    else{
      d_id[row*n+col] /= d_a[row*n+row];
    }
  }
}

__global__ void reduce_row(double *d_1, double *d_2, int n, int current_row, bool reduce_id){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if( col<n && row< n){
    if (row != current_row){
      if (col != current_row || reduce_id){
        d_1[row*n+col] -= d_1[current_row*n+col]*d_2[row*n+current_row];
      }
    }
  }
}

__global__ void add_kernel(double *d_a, double *d_b, double *d_c, int n, int m){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if( col < m && row < n){
    d_c[row + m*col] = d_a[row + m*col] + d_b[row + m*col];
  }
}

__global__ void subtract_kernel(double *d_a, double *d_b, double *d_c, int n, int m){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if( col < m && row < n){
    d_c[row + m*col] = d_a[row + m*col] - d_b[row + m*col];
  }
}

__global__ void multiply_kernel(double *d_a, double *d_b, double *d_c, int m, int n, int k){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float tmp = 0;
  if(row < m && col < k){
    for(int i = 0; i < n; i++){
      tmp += d_a[row * m + i] * d_b[i * k + col];
    }
    d_c[row * k + col] = tmp;
  }
}

GPUToolkit::GPUToolkit(){

}

GPUToolkit::~GPUToolkit(){

}

Matrix GPUToolkit::add(Matrix A, Matrix B){
  int A_n  = A.getN();
  int A_m  = A.getM();
  int B_n  = B.getN();
  int B_m  = B.getM();
  if ( A_n != B_n || A_m != B_m ){
    throw "How am I supposed to add a physicist to a humanist?!";
  }
  Matrix C(A_n, A_m);
  double *h_a = A.getArrayPointer();
  double *h_b = B.getArrayPointer();
  double *h_c = C.getArrayPointer();
  // Allocate memory space on the device
  double *d_a;
  double *d_b;
  double *d_c;
  hipMalloc((void **)&d_a, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_b, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_c, A_n*A_m*sizeof(double));
  // copy matrices from host to device memory
  hipMemcpy(d_a, h_a, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  // set grid
  dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid2( (A_n+dimBlock2.x-1)/dimBlock2.x,(A_n+dimBlock2.y-1)/dimBlock2.y );
  // launch kernel
  add_kernel <<<dimGrid2, dimBlock2>>> (d_a, d_b, d_c, A_n, A_m);
  // copy result to host
  hipMemcpy(h_c, d_c, A_n*A_m*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

  return C;
}

Matrix GPUToolkit::subtract(Matrix A, Matrix B){
  int A_n  = A.getN();
  int A_m  = A.getM();
  int B_n  = B.getN();
  int B_m  = B.getM();
  if ( A_n != B_n || A_m != B_m ){
    throw "How am I supposed to subtract a humanist from a physicist?!";
  }
  Matrix C(A_n, A_m);
  double *h_a = A.getArrayPointer();
  double *h_b = B.getArrayPointer();
  double *h_c = C.getArrayPointer();
  // Allocate memory space on the device
  double *d_a;
  double *d_b;
  double *d_c;
  hipMalloc((void **)&d_a, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_b, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_c, A_n*A_m*sizeof(double));
  // copy matrices from host to device memory
  hipMemcpy(d_a, h_a, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  // set grid
  dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid2( (A_n+dimBlock2.x-1)/dimBlock2.x,(A_n+dimBlock2.y-1)/dimBlock2.y );
  // launch kernel
  subtract_kernel <<<dimGrid2, dimBlock2>>> (d_a, d_b, d_c, A_n, A_m);
  // copy result to host
  hipMemcpy(h_c, d_c, A_n*A_m*sizeof(double), hipMemcpyDeviceToHost);
  // be a hero and clean after yourself!
  hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

  return C;
}

Matrix GPUToolkit::multiply(Matrix A, Matrix B){
  int A_n  = A.getN();
  int A_m  = A.getM();
  int B_n  = B.getN();
  int B_m  = B.getM();
  if ( A_m != B_n ){
    throw "Matrix multiplication is defined only for matrices with the same inner dimensions!";
  }
  Matrix C(A_n, B_m);
  double *h_a = A.getArrayPointer();
  double *h_b = B.getArrayPointer();
  double *h_c = C.getArrayPointer();
  // Allocate memory space on the device
  double *d_a;
  double *d_b;
  double *d_c;
  hipMalloc((void **)&d_a, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_b, A_n*A_m*sizeof(double));
  hipMalloc((void **)&d_c, A_n*A_m*sizeof(double));
  // copy matrices from host to device memory
  hipMemcpy(d_a, h_a, A_n*A_m*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, A_m*B_n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, A_n*B_m*sizeof(double), hipMemcpyHostToDevice);
  // set grid
  dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid2( (A_n+dimBlock2.x-1)/dimBlock2.x,(A_n+dimBlock2.y-1)/dimBlock2.y );
  // launch kernel
  multiply_kernel <<<dimGrid2, dimBlock2>>> (d_a, d_b, d_c, A_n, A_m, B_m);
  // copy result to host
  hipMemcpy(h_c, d_c, A_n*A_m*sizeof(double), hipMemcpyDeviceToHost);
  // be a hero and clean after yourself!
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return C;
}

Matrix GPUToolkit::findInverse(Matrix A){
  int A_n = A.getN();
  int A_m = A.getM();
  if ( A_n != A_m ){
    throw "Inverse matrix is undefined for non-square matrices!";
  }
  Matrix Id(A_n,A_n);
  Id = 1;
  double *h_a = A.getArrayPointer();
  double *h_id = Id.getArrayPointer();

  // Allocate memory space on the device
  double *d_a;
  double *d_id;
  hipMalloc((void **)&d_a, A_n*A_n*sizeof(double));
  hipMalloc((void **)&d_id, A_n*A_n*sizeof(double));

  // copy matrices from host to device memory
  hipMemcpy(d_a, h_a, A_n*A_n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_id, h_id, A_n*A_n*sizeof(double), hipMemcpyHostToDevice);

  // iterate over matrix rows
  for(int row=0; row<A_n; row++){
    //set dimensions
    dim3 dimBlock(BLOCK_SIZE*BLOCK_SIZE, 1);
    dim3 dimGrid( (A_n+dimBlock.x-1)/dimBlock.x,1);
    find_pivots <<< dimGrid, dimBlock>>> (d_a, d_id, A_n, row);
    normalize_row <<< dimGrid, dimBlock>>> (d_a, d_id, A_n, row);
    // change grid dimensions; row reduction can be done at once on 2D grid
    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2( (A_n+dimBlock2.x-1)/dimBlock2.x,(A_n+dimBlock2.y-1)/dimBlock2.y );
    reduce_row <<<dimGrid2, dimBlock2>>> (d_id, d_a, A_n, row, true);
    reduce_row <<<dimGrid2, dimBlock2>>> (d_a, d_a, A_n, row, false);
  }
  hipMemcpy(h_a, d_a, A_n*A_n*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(h_id, d_id, A_n*A_n*sizeof(double), hipMemcpyDeviceToHost);

  // be a hero and clean after yourself!
  hipFree(d_a);
	hipFree(d_id);
  return Id;
}
